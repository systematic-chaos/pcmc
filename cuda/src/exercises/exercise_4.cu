/*
 * CUDA program that computes the product of a matrix of dimensions `m * n`
 * and a vector with `n` real elements expressed in simple precision.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA Kernel for the Device

// Main program in the Host
int main(void) {
    // TODO

    // Reset the Device and exit
    hipDeviceReset();
    return 0;
}
